#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__global__ void kernalFindKey(int N, int n, int g, int h, int p){
/*int nthreads = modExp(2,n,p);
int blockid = //No clue as to what this would be however need help;
int Nblock = nthreads/1024;*/
if (modExp(g, blockIdx.x + 1, p) == h)
    d_x = blockIdx.x + 1;
//convert this to only 1 if statement.
/*if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {   
      if (modExp(g,i+1,p)==h) {
//        printf("Secret key found! x = %u \n", i+1);
        x=i+1; 
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
*/
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

//declare storage for an ElGamal cryptosytem
  unsigned int N = atoi(argv[1]);
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);

  printf("Reading file.\n");
  

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  /* Q3 After finding the secret key, decrypt the message */
  FILE* Pkey = fopen("public_key.txt", "r");
  unsigned int *data_key = (unsigned int*) malloc(4*sizeof(unsigned int));
  for (unsigned int i = 0; i < 4; i++)
      fscanf(Pkey, "%u", data_key + i);
  fclose(Pkey);
  n = data_key[0];
  p = data_key[1];
  g = data_key[2];
  h = data_key[3];
  printf("%u, %u, %u, %u\n", n,p,g,h);
    // find the secret key
  
  //Idea is to make host storage so as to pass info to the device storage.
  unsigned int h_a, h_b, h_c;
  h_n = (unsigned int *) malloc(N*sizeof(unsigned int));
  h_g = (unsigned int *) malloc(N*sizeof(unsigned int));
  h_p = (unsigned int *) malloc(N*sizeof(unsigned int));  
  h_h = (unsigned int *) malloc(N*sizeof(unsigned int));


  size_t inputMem = 2* *sizeof(double);//missing a number
  size_t outMem = *sizeof(double);//missing a number  

  unsigned int d_n, d_g, d_h, d_p;
  hipMalloc(&d_n, N*sizeof(unsigned int));
  hipMalloc(&d_g, N*sizeof(unsigned int)); 
  hipMalloc(&d_p, N*sizeof(unsigned int));
  hipMalloc(&d_h, N*sizeof(unsigned int));

  hipMemcpy(d_n, h_n, N*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_g, h_g, N*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_p, h_p, N*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(d_h, h_h, N*sizeof(unsigned int), hipMemcpyHostToDevice);
 
  unsigned int Nthreads = modExp(2, N, p);
  unsigned int Nblocks = Nthreads/1024;
  unsigned int x;
  x = kernalFindKey<<<Nblocks, Nthreads>>> (N, d_n, d_g, d_p, d_h);
  hipDeviceSynchronize();
  
  hipFree(d_n);
  hipFree(d_g);
  hipFree(d_p);
  hipFree(d_h);

  free(h_n);
  free(h_g);
  free(h_p);
  free(h_h);

  /* Q3 After finding the secret key, decrypt the message */
  FILE* message = fopen("message.txt" , "r"); 
  unsigned int *m_array , *a_array;
  fscanf(message, "%u", &Nints);
  printf("Nints is %u\n", Nints);
  m_array = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  a_array = (unsigned int*) malloc(Nints*sizeof(unsigned int));
    
  // fscanf(message, "%u \n", &Nints);
  unsigned char *data_message = (unsigned char*) malloc(Nints*sizeof(unsigned char));
  
  printf("Nints is %u\n", Nints);
  for (unsigned int k = 0; k < Nints; k++)
  {
	fscanf(message, "%u %u", m_array+k, a_array+k);
        
        printf("%u is m\n", m_array[k]);
        printf("%u is a\n", a_array[k]);
  }
  fclose(message);  
  int bufferSize = 1024;
  unsigned char *message2 = (unsigned char*) malloc(bufferSize*sizeof(unsigned char));
  unsigned int Nchars = ((n-1)/8) * Nints;
  ElGamalDecrypt(m_array, a_array, Nints, p, x);
  convertZToString(m_array, Nints, message2, Nchars);
  printf("Decrypted Message = \"%s\"\n", message2);
  return 0;
}
