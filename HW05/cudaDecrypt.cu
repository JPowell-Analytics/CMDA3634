#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

__device__ unsigned int modprodCuda(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

__device__ unsigned int modExpCuda(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprodCuda(aExpb, z, p);
    z = modprodCuda(z, z, p);
    b /= 2;
  }
  return aExpb;
}

__global__ void kernalFindKey(int p, int g, int h, int device_array){
/*int nthreads = modExp(2,n,p);
int blockid = //No clue as to what this would be however need help;
int Nblock = nthreads/1024;*/

unsigned int d_x, threadId, blockId, Nblock;
threadId = threadIdx.x;
blockId = blockIdx.x;
Nblock = blockDim.x;
unsigned  int id = threadId + Nblock*blockId;
if (id < p-1){
	if (modExpCuda(g, id, p) == h)
    		device_array[0] = id;
}

//convert this to only 1 if statement.
/*if (x==0 || modExp(g,x,p)!=h) {
    printf("Finding the secret key...\n");
    double startTime = clock();
    for (unsigned int i=0;i<p-1;i++) {   
      if (modExp(g,i+1,p)==h) {
//        printf("Secret key found! x = %u \n", i+1);
        x=i+1; 
      } 
    }
    double endTime = clock();

    double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
    double work = (double) p;
    double throughput = work/totalTime;

    printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
*/
}

int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

//declare storage for an ElGamal cryptosytem
  //unsigned int N = atoi(argv[1]);
  unsigned int n, p, g, h, x;
  unsigned int Nints;

  //get the secret key from the user
/*  printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  char stat = scanf("%u",&x);
*/
  printf("Reading file.\n");
  

  /* Q3 Complete this function. Read in the public key data from public_key.txt
    and the cyphertexts from messages.txt. */

  /* Q3 After finding the secret key, decrypt the message */
  FILE* Pkey = fopen("public_key.txt", "r");
  unsigned int *data_key = (unsigned int*) malloc(4*sizeof(unsigned int));
  for (unsigned int i = 0; i < 4; i++)
      fscanf(Pkey, "%u", data_key + i);
  fclose(Pkey);
  n = data_key[0];
  p = data_key[1];
  g = data_key[2];
  h = data_key[3];
  printf("%u, %u, %u, %u\n", n,p,g,h);
    // find the secret key

 FILE* message = fopen("message.txt" , "r"); 
  unsigned int *m_array , *a_array;
  fscanf(message, "%u", &Nints);
  printf("Nints is %u\n", Nints);
  m_array = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  a_array = (unsigned int*) malloc(Nints*sizeof(unsigned int));
    
  // fscanf(message, "%u \n", &Nints);
  unsigned char *data_message = (unsigned char*) malloc(Nints*sizeof(unsigned char));
  
  printf("Nints is %u\n", Nints);
  for (unsigned int k = 0; k < Nints; k++)
  {
	fscanf(message, "%u %u", m_array+k, a_array+k);
  }
  fclose(message);

  unsigned int Nthreads = 32;
  unsigned int *device_array, *host_array;
  host_array = (unsigned int *) malloc(Nthreads*sizeof(unsigned int));
  dim3 in(Nthreads, 1, 1);
  dim3 out((p+Nthreads-1)/Nthreads, 1, 1);
  hipMalloc(&device_array, Nthreads*sizeof(unsigned int)); 
  
  kernalFindKey<<<out, in>>> (p, g, h, device_array);
  hipDeviceSynchronize();
  cudaMemCpy(host_array, device_array, Nthreads*sizeof(unsigned int), hipMemcpyDeviceToHost);
  x = host_array;
  hipFree(device_array);

  free(host_array);
  
  /* Q3 After finding the secret key, decrypt the message */
  
  
  int bufferSize = 1024;
  unsigned char *message2 = (unsigned char*) malloc(bufferSize*sizeof(unsigned char));
  unsigned int Nchars = ((n-1)/8) * Nints;

  ElGamalDecrypt(m_array, a_array, Nints, p, x);
  convertZToString(m_array, Nints, message2, Nchars);
  printf("Decrypted Message = \"%s\"\n", message2);
  return 0;
}
